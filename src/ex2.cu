#include "hip/hip_runtime.h"
#include "ex2.h"
#include <cuda/atomic>

#define HISTOGRAM_SIZE 256
#define NUM_OF_THREADS 1024
#define WRAP_SIZE 32

__device__ void prefix_sum(int arr[], int arr_size) {
    // TODO complete according to hw1
    int tid = threadIdx.x;
    int increment;
    for (int stride = 1; stride < arr_size; stride *= 2) {
    if (tid >= stride && tid < arr_size) {
        increment = arr[tid - stride];
    }
    __syncthreads();
    if (tid >= stride && tid < arr_size) {
        arr[tid] += increment;
    }
    __syncthreads();
    }
}

/**
 * Perform interpolation on a single image
 *
 * @param maps 3D array ([TILES_COUNT][TILES_COUNT][256]) of    
 *             the tiles’ maps, in global memory.
 * @param in_img single input image, in global memory.
 * @param out_img single output buffer, in global memory.
 */
__device__
 void interpolate_device(uchar* maps ,uchar *in_img, uchar* out_img);

__device__
void process_image(uchar *in, uchar *out, uchar* maps) {
    // TODO complete according to hw1
    int ti = threadIdx.x;
    int tg = ti / TILE_WIDTH;
    int bi = blockIdx.x;
    int workForThread = (TILE_WIDTH * TILE_WIDTH) / NUM_OF_THREADS; // in bytes
    uchar imageVal;

    __shared__ int sharedHist[HISTOGRAM_SIZE]; // maybe change to 16 bit ? will be confilcits on same bank 

    int imageStartIndex = 0;// bi * IMG_HEIGHT * IMG_WIDTH;
    int mapStartIndex = 0;// bi * TILE_COUNT * TILE_COUNT * HISTOGRAM_SIZE;
    int tileStartIndex;
    int insideTileIndex;
    int curIndex;
    for (int i = 0 ; i < TILE_COUNT * TILE_COUNT; i++)
    {
        // calc tile index in image buffer (shared between al threads in block)
        tileStartIndex = imageStartIndex + i % TILE_COUNT * TILE_WIDTH + (i / TILE_COUNT) * (TILE_WIDTH *TILE_WIDTH) * TILE_COUNT;
        // zero shared buffer histogram values
        if (ti < 256)
        {
            sharedHist[ti] = 0;
        }
        __syncthreads();
       for (int j = 0; j < workForThread; j++)
            {
                // calc index in tile buffer for each thread
                insideTileIndex = tg * TILE_WIDTH * TILE_COUNT + ti % TILE_WIDTH + (NUM_OF_THREADS / TILE_WIDTH) * TILE_WIDTH * TILE_COUNT * j;
                // sum tile index and index inside tile to find relevant byte for thread in cur iteration
                curIndex = tileStartIndex + insideTileIndex;
                // update histogram
                imageVal = in[curIndex];
                atomicAdd(sharedHist + imageVal, 1);
        }
    
        __syncthreads();
        
        // calc CDF using prefix sumpwdon histogram buffer

        prefix_sum(sharedHist, HISTOGRAM_SIZE);

        __syncthreads();
        // calc map value for each index
        if (ti < 256)
        {
            maps[mapStartIndex + HISTOGRAM_SIZE * i + ti] = (float(sharedHist[ti]) * 255)  / (TILE_WIDTH * TILE_WIDTH);
        }
    }

    __syncthreads();
    // interpolate image using given maps buffer
    interpolate_device(maps + mapStartIndex, in + imageStartIndex, out + imageStartIndex);
    return; 
}

__global__
void process_image_kernel(uchar *in, uchar *out, uchar* maps){
    process_image(in, out, maps);
}

struct Stream_Wrap
{
    hipStream_t stream;
    int streamImageId;
    uchar *taskMaps;
    uchar *imgIn;
    uchar *imgOut;
};

class streams_server : public image_processing_server
{
private:
    // TODO define stream server context (memory buffers, streams, etc...)
    Stream_Wrap streams[STREAM_COUNT];

public:
    streams_server()
    {
        // TODO initialize context (memory buffers, streams, etc...)
        for (int i = 0; i < STREAM_COUNT; i++) {
            CUDA_CHECK(hipStreamCreate(&streams[i].stream));
            streams[i].streamImageId = -1; // avialble
            CUDA_CHECK(hipMalloc((void**)&streams[i].taskMaps, TILE_COUNT * TILE_COUNT * HISTOGRAM_SIZE));
            CUDA_CHECK(hipMalloc((void**)&streams[i].imgIn,  IMG_WIDTH * IMG_HEIGHT));
            CUDA_CHECK(hipMalloc((void**)&streams[i].imgOut,IMG_WIDTH * IMG_HEIGHT));
        }
    }

    ~streams_server() override
    {
        // TODO free resources allocated in constructor
        for (int i = 0; i < STREAM_COUNT; i++) {
            CUDA_CHECK(hipStreamDestroy(streams[i].stream));
            CUDA_CHECK(hipFree(streams[i].taskMaps));
            CUDA_CHECK(hipFree(streams[i].imgIn));
            CUDA_CHECK(hipFree(streams[i].imgOut));
        }
    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        // TODO place memory transfers and kernel invocation in streams if possible.
        for (int i = 0; i < STREAM_COUNT; i++)
        {
            if (streams[i].streamImageId == -1)
            {
                streams[i].streamImageId = img_id;
                CUDA_CHECK(hipMemcpyAsync(streams[i].imgIn, img_in , IMG_WIDTH * IMG_HEIGHT,hipMemcpyHostToDevice, streams[i].stream));
                process_image_kernel<<<1, NUM_OF_THREADS, 0, streams[i].stream>>>(img_in, streams[i].imgOut, streams[i].taskMaps);
                CUDA_CHECK(hipMemcpyAsync(img_out, streams[i].imgOut, IMG_WIDTH * IMG_HEIGHT, hipMemcpyDeviceToHost, streams[i].stream));
                return true;
            }
        }
        return false;
    }

    bool dequeue(int *img_id) override
    {
        // TODO query (don't block) streams for any completed requests.
        for (int i = 0; i < STREAM_COUNT; i++)
        {
            if (streams[i].streamImageId != -1)
            {
                hipError_t status = hipStreamQuery(streams[i].stream); // TODO query diffrent stream each iteration
                switch (status) {
                case hipSuccess:
                    // TODO return the img_id of the request that was completed.
                    // printf("bla");
                    *img_id = streams[i].streamImageId;
                    streams[i].streamImageId = -1;
                    return true;
                case hipErrorNotReady:
                    return false;
                default:
                    CUDA_CHECK(status);
                    return false;
                }
            }
            
        }

        return false;
    }
};

std::unique_ptr<image_processing_server> create_streams_server()
{
    return std::make_unique<streams_server>();
}




// TODO implement a lock
// TODO implement a MPMC queue
// TODO implement the persistent kernel
// TODO implement a function for calculating the threadblocks count

class queue_server : public image_processing_server
{
private:
    // TODO define queue server context (memory buffers, etc...)
public:
    queue_server(int threads)
    {
        // TODO initialize host state
        // TODO launch GPU persistent kernel with given number of threads, and calculated number of threadblocks
    }

    ~queue_server() override
    {
        // TODO free resources allocated in constructor
    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        // TODO push new task into queue if possible
        return false;
    }

    bool dequeue(int *img_id) override
    {
        // TODO query (don't block) the producer-consumer queue for any responses.
        return false;

        // TODO return the img_id of the request that was completed.
        //*img_id = ... 
        return true;
    }
};

std::unique_ptr<image_processing_server> create_queues_server(int threads)
{
    return std::make_unique<queue_server>(threads);
}
